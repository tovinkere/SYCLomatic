
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t upper_lower, int n,
          float **a, int lda, int *info, int group_count) {
  // Start
  hipsolverDnSpotrfBatched(handle /*cusolverDnHandle_t*/,
                          upper_lower /*cublasFillMode_t*/, n /*int*/,
                          a /*float ***/, lda /*int*/, info /*int **/,
                          group_count /*int*/);
  // End
}
